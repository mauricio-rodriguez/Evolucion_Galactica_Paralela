#include "hip/hip_runtime.h"
#define __CUDA
#include <cstdio>
// #include <iostream>
#include <cutil.h>
#include "hermite4-gpu.h"

#define NTHREADS 128

// #define NJBLOCKS 16
// #define NJBLOCKS_ 16

#define NJBLOCKS  28 // GTX470
#define NJBLOCKS_ 32

#define NREDUCE (NTHREADS/NJBLOCKS_)
#define NIBLOCKS 32
#define NIMAX (NTHREADS * NIBLOCKS) // 2048
#define GPU_REDUCE

struct Force_dev{
	float2 acc[3];
	// float2 pot;
	float jrk[3];
	// float pad0;
	float pad[3];
	__device__ Force_dev(){
		// acc[0] = acc[1] = acc[2] = pot = make_float2(0.f, 0.f);
		acc[0] = acc[1] = acc[2] = make_float2(0.f, 0.f);
		jrk[0] = jrk[1] = jrk[2] =  0.f;
	}
};

__device__ void force_reduce(Force_dev &fl, Force_dev &fr){
#pragma unroll
	for(int k=0; k<3; k++){
		fl.acc[k] = float2_accum(fl.acc[k], fr.acc[k].x);
		fl.acc[k] = float2_accum(fl.acc[k], fr.acc[k].y);
		fl.jrk[k] += fr.jrk[k];
	}
	// fl.pot = float2_accum(fl.pot, fr.pot.x);
	// fl.pot = float2_accum(fl.pot, fr.pot.y);
}

__device__ void h4_kernel(
		const Predictor &ip,
		const Predictor &jp,
		Force_dev &fo,
		float eps2){
#if 0
	float dx = float2_sub(jp.pos[0], ip.pos[0]);
	float dy = float2_sub(jp.pos[1], ip.pos[1]);
	float dz = float2_sub(jp.pos[2], ip.pos[2]);

	float dvx = jp.vel[0] - ip.vel[0];
	float dvy = jp.vel[1] - ip.vel[1];
	float dvz = jp.vel[2] - ip.vel[2];

	float dax = jp.acc[0] - ip.acc[0];
	float day = jp.acc[1] - ip.acc[1];
	float daz = jp.acc[2] - ip.acc[2];
#else
	float dx = (jp.posH.x - ip.posH.x) + (jp.posL.x - ip.posL.x);
	float dy = (jp.posH.y - ip.posH.y) + (jp.posL.y - ip.posL.y);
	float dz = (jp.posH.z - ip.posH.z) + (jp.posL.z - ip.posL.z);

	float dvx = jp.vel.x - ip.vel.x;
	float dvy = jp.vel.y - ip.vel.y;
	float dvz = jp.vel.z - ip.vel.z;
#endif

	float r2 = eps2 + dx*dx + dy*dy + dz*dz;
	float drdv =  dx*dvx +  dy*dvy +  dz*dvz;

	float rinv1 = rsqrtf(r2);
	float rinv2 = rinv1 * rinv1;
	float alpha = (drdv)*rinv2;
	// rinv1 *= jp.mass;
	rinv1 *= jp.posH.w;
	float rinv3 = rinv1 * rinv2;

	// float pot = rinv1;
	float ax = rinv3*dx;
	float ay = rinv3*dy;
	float az = rinv3*dz;
	float jx = rinv3*dvx + (-3.f*alpha)*ax;
	float jy = rinv3*dvy + (-3.f*alpha)*ay;
	float jz = rinv3*dvz + (-3.f*alpha)*az;

#if 0
	if(r2 != eps2){
		fo.pot = float2_accum(fo.pot, pot);
	}
#endif
	fo.acc[0] = float2_accum(fo.acc[0], ax);
	fo.acc[1] = float2_accum(fo.acc[1], ay);
	fo.acc[2] = float2_accum(fo.acc[2], az);
	fo.jrk[0] += jx;
	fo.jrk[1] += jy;
	fo.jrk[2] += jz;
}

__global__ void h4_gravity(
		int ni,
		int nj,
		Predictor ipred[],
		Predictor jpred[],
		Force_dev force[][NJBLOCKS_],
		float eps2){
	int ibid = blockIdx.x;
	int jbid = blockIdx.y;
	int tid = threadIdx.x;
	int iaddr = tid + NTHREADS * ibid;
	int jstart = (nj * (jbid  )) / NJBLOCKS;
	int jend   = (nj * (jbid+1)) / NJBLOCKS;

	// small kernel opt
	int nskip = 1;
	int niloc = ni - NTHREADS * ibid;
	if(niloc <= NTHREADS/2) nskip = 2;
	if(niloc <= NTHREADS/4) nskip = 4;
	if(niloc <= NTHREADS/8) nskip = 8;
	if(niloc <= NTHREADS/16) nskip = 16;
	if(niloc <= NTHREADS/32) nskip = 32;
	int joff = tid / (NTHREADS/nskip);

	__shared__ Predictor jpshare[NTHREADS];
	Force_dev fo;
	Predictor ip = ipred[tid % (NTHREADS/nskip) + NTHREADS * ibid];
	for(int j=jstart; j<jend; j+=NTHREADS){
		__syncthreads();
#if 0
		jpshare[tid] = jpred[j+tid];
#else
		float4 *src = (float4 *)&jpred[j];
		float4 *dst = (float4 *)jpshare;
		for(int it=0; it<sizeof(Predictor)/sizeof(float4); it++){
			dst[tid] = src[tid];
			dst += NTHREADS;
			src += NTHREADS;
		}
#endif
		__syncthreads();
		if(jend-j < NTHREADS){
			for(int jj=0; jj<jend-j; jj+=nskip){
				Predictor &jp = jpshare[jj+joff];
				if(jj+joff < jend-j) h4_kernel(ip, jp, fo, eps2);
			}
		}else{
#if 0
#pragma unroll
			for(int jj=0; jj<NTHREADS; jj+=nskip){
				Predictor &jp = jpshare[jj+joff];
				h6_kernel(ip, jp, fo, eps2);
			}
#else
			for(int jj=0; jj<NTHREADS; jj+=4*nskip){
				Predictor &jp0 = jpshare[0*nskip+jj+joff];
				Predictor &jp1 = jpshare[1*nskip+jj+joff];
				Predictor &jp2 = jpshare[2*nskip+jj+joff];
				Predictor &jp3 = jpshare[3*nskip+jj+joff];
				h4_kernel(ip, jp0, fo, eps2);
				h4_kernel(ip, jp1, fo, eps2);
				h4_kernel(ip, jp2, fo, eps2);
				h4_kernel(ip, jp3, fo, eps2);
			}
#endif
		}
	}
	// horizontal reduce
	// __shared__ Force_dev foshare[NTHREADS];
	Force_dev *foshare = (Force_dev *)jpshare;
	__syncthreads();
	foshare[tid] = fo;
	__syncthreads();
	if(nskip > 1){
		if(tid < NTHREADS/2){
			force_reduce(foshare[tid], foshare[tid + NTHREADS/2]);
		}
		__syncthreads();
	}
	if(nskip > 2){
		if(tid < NTHREADS/4){
			force_reduce(foshare[tid], foshare[tid + NTHREADS/4]);
		}
		__syncthreads();
	}
	if(nskip > 4){
		if(tid < NTHREADS/8){
			force_reduce(foshare[tid], foshare[tid + NTHREADS/8]);
		}
		__syncthreads();
	}
	if(nskip > 8){
		if(tid < NTHREADS/16){
			force_reduce(foshare[tid], foshare[tid + NTHREADS/16]);
		}
		__syncthreads();
	}
	if(nskip > 16){
		if(tid < NTHREADS/32){
			force_reduce(foshare[tid], foshare[tid + NTHREADS/32]);
		}
		__syncthreads();
	}
	// store
	if(tid < niloc){
		fo = foshare[tid];
		force[iaddr][jbid] = fo;
	}
}

#ifdef GPU_REDUCE
__global__ void reduce_kernel(
		Force_dev fo_dev[][NJBLOCKS_],
		Force_dev fo_reduce[])
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int ioff = bid * NREDUCE;
#if 0
	__shared__ Force_dev fo_share[NTHREADS];
#else
	__shared__ Predictor jpshare[NTHREADS];
	Force_dev *fo_share = (Force_dev *)jpshare;
#endif
#if 0
	fo_share[tid] = fo_dev[ioff][tid];
#else
	float4 *src = (float4 *)fo_dev[ioff];
	float4 *dst = (float4 *)fo_share;
	for(int it=0; it<sizeof(Force_dev)/sizeof(float4); it++){
		dst[tid] = src[tid];
		dst += NTHREADS;
		src += NTHREADS;
	}
#endif
	__syncthreads();

	int n = NJBLOCKS_;
	while(n > 1){
		n /= 2;
		if(tid % NJBLOCKS_ < n){
			force_reduce(fo_share[tid], fo_share[tid + n]);
		}
	}
	__syncthreads();

	if(tid % NJBLOCKS_ == 0){
		// fo_reduce[ioff + tid / NJBLOCKS_] = fo_share[tid];
		fo_share[tid / NJBLOCKS_] = fo_share[tid];
	}
	__syncthreads();
#if 0
	if(tid < NREDUCE){
		fo_reduce[ioff + tid] = fo_share[tid];
	}
#else
	if(tid < NREDUCE * sizeof(Force_dev) / sizeof(float)){ // (tid < 96)
		float *dst = (float *)&fo_reduce[ioff];
		float *src = (float *)fo_share;
		dst[tid] = src[tid];
	}
#endif
}
#endif

extern double wtime();

void calc_force(
		int nitot, 
		int nj, 
		float eps2,
		Predictor ipred[],
		Predictor jpred[],
		Force     force[],
		double &t1,
		double &t_send,
		double &t_recv){
	static Predictor *jp_dev = NULL;
	static Predictor *ip_dev = NULL;
	static Force_dev (*fo_dev)[NJBLOCKS_] = NULL;
#ifdef GPU_REDUCE
	static Force_dev (*fo_reduce) = NULL;
	static Force_dev (*fo_host) = NULL;
#else
	static Force_dev (*fo_host)[NJBLOCKS_] = NULL;
#endif

	if(jp_dev == NULL){ // first call
		/*
		   const int dev = 0;
		   CUDA_SAFE_CALL(hipSetDevice(dev));
		   hipDeviceProp_t deviceProp;
		   CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
		   printf("GPU: %s\n", deviceProp.name);
		 */
		hipMalloc((void **)&jp_dev, (nj + NTHREADS) * sizeof(Predictor));
		hipMalloc((void **)&ip_dev, NIMAX * sizeof(Predictor));
		hipMalloc((void **)&fo_dev, NIMAX * sizeof(*fo_dev));
		hipMemset(fo_dev, 0, NIMAX * sizeof(*fo_dev));
#ifdef GPU_REDUCE
		hipMalloc((void **)&fo_reduce, NIMAX * sizeof(*fo_reduce));
#endif
		hipHostMalloc((void **)&fo_host, NIMAX * sizeof(*fo_host));
	}

	hipMemcpy(jp_dev, jpred, nj * sizeof(Predictor), hipMemcpyHostToDevice);
	t1 = wtime();

	int nimax = NIMAX;
	for(int ioff=0; ioff<nitot; ioff+=nimax){
		int ni = std::min(nimax, nitot-ioff);
		double t2 = wtime();
		hipMemcpy(ip_dev, ipred+ioff, ni * sizeof(Predictor), hipMemcpyHostToDevice);
		double t3 = wtime();
		t_send += t3 - t2;
		// kernel call
		int niblocks = 1 + (ni-1) / NTHREADS;
		dim3 grid(niblocks, NJBLOCKS, 1);
		dim3 threads(NTHREADS, 1, 1);
		// std::cerr << "call h6_gravity " << niblocks << std::endl;
		//int sharedMemSize = NTHREADS * sizeof(Predictor);
		// h6_gravity <<< grid, threads, sharedMemSize >>>
		//	(ni, nj, ip_dev, jp_dev, fo_dev, eps2);
		h4_gravity <<< grid, threads >>>
			(ni, nj, ip_dev, jp_dev, fo_dev, eps2);

#ifdef GPU_REDUCE
		dim3 grid_reduce(1 + (ni-1)/NREDUCE, 1, 1);
		reduce_kernel <<< grid_reduce, threads >>> (fo_dev, fo_reduce);
		hipDeviceSynchronize();
		double t4 = wtime();
		hipMemcpy(fo_host, fo_reduce, ni * sizeof(*fo_reduce), hipMemcpyDeviceToHost);
		double t5 = wtime();
		t_recv += t5 - t4;
		for(int i=0; i<ni; i++){
			Force f; // 0 flashed by the constructer
			Force_dev &fo = fo_host[i];
			f.acc.x = float2_reduce(fo.acc[0]);
			f.acc.y = float2_reduce(fo.acc[1]);
			f.acc.z = float2_reduce(fo.acc[2]);
			// f.pot   = float2_reduce(fo.pot);
			f.jrk.x = fo.jrk[0];
			f.jrk.y = fo.jrk[1];
			f.jrk.z = fo.jrk[2];
			force[ioff + i] = f;
		}
#else
		hipMemcpy(fo_host, fo_dev, ni * sizeof(*fo_dev), hipMemcpyDeviceToHost);
		// std::cerr << "done" << std::endl;
		for(int i=0; i<ni; i++){
			Force f; // 0 flashed by the constructer
			for(int jb=0; jb<NJBLOCKS; jb++){
				Force_dev &fo = fo_host[i][jb];
				f.acc.x += float2_reduce(fo.acc[0]);
				f.acc.y += float2_reduce(fo.acc[1]);
				f.acc.z += float2_reduce(fo.acc[2]);
				f.pot   -= float2_reduce(fo.pot);
				f.jrk.x += fo.jrk[0];
				f.jrk.y += fo.jrk[1];
				f.jrk.z += fo.jrk[2];
				f.snp.x += fo.snp[0];
				f.snp.y += fo.snp[1];
				f.snp.z += fo.snp[2];
			}
			force[ioff + i] = f;
		}
#endif
	}
}

__global__ void pot_kernel(
		int js,
		int je,
		float eps2,
		Posm posm[],
		float2 pot[]){
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int iaddr = tid + NTHREADS * bid;
	Posm ip = posm[iaddr];
	float2 poti = make_float2(0.f, 0.f);
	for(int j=js; j<je; j+=NTHREADS){
		__shared__ Posm posmshare[NTHREADS];
		__syncthreads();
		posmshare[tid] = posm[j + tid];
		__syncthreads();
		int njj = NTHREADS < je-j ? NTHREADS : je-j;
		for(int jj=0; jj< njj; jj++){
			Posm &jp = posmshare[jj];
			float dx = float2_sub(jp.pos[0], ip.pos[0]);
			float dy = float2_sub(jp.pos[1], ip.pos[1]);
			float dz = float2_sub(jp.pos[2], ip.pos[2]);
			float r2 = eps2 + dx*dx + dy*dy + dz*dz;
			float mrinv = jp.mass * rsqrtf(r2);
			if(r2 > eps2) poti = float2_accum(poti, mrinv);
		}
	}
	pot[iaddr] = poti;
}

void calc_pot(
		int ni,
		int js,
		int je,
		float eps2,
		Posm posm[],
		double dpot[]){
	Posm *posm_dev;
	float2 *pot, *pot_dev;
	hipMalloc((void **)&posm_dev, (ni+NTHREADS) * sizeof(Posm));
	hipMalloc((void **)&pot_dev, (ni+NTHREADS) * sizeof(float2));
	hipHostMalloc((void **)&pot, (ni+NTHREADS) * sizeof(float2));

	hipMemcpy(posm_dev, posm, ni * sizeof(Posm), hipMemcpyHostToDevice);

	int nblocks = 1 + (ni-1) / NTHREADS;
	dim3 grid(nblocks, 1, 1);
	dim3 threads(NTHREADS, 1, 1);
	int sharedMemSize = NTHREADS * sizeof(Posm);

	pot_kernel <<< grid, threads, sharedMemSize >>>
		(js, je, eps2, posm_dev, pot_dev);

	hipMemcpy(pot, pot_dev, ni * sizeof(float2), hipMemcpyDeviceToHost);
	for(int i=0; i<ni; i++){
		dpot[i] = -float2_reduce(pot[i]);
	}

	hipFree(posm_dev);
	hipFree(pot_dev);
	hipHostFree(pot);
}

void CUDA_MPI_Init(int myRank){
	int numGPU;
	CUDA_SAFE_CALL(hipGetDeviceCount(&numGPU));
	const int dev = myRank % numGPU;
	CUDA_SAFE_CALL(hipSetDevice(dev));
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
	printf("Rank %02d : GPU %d : %s\n", myRank, dev, deviceProp.name); 

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(h4_gravity),    hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(reduce_kernel), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(pot_kernel),    hipFuncCachePreferShared);
}
