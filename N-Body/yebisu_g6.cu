#include "hip/hip_runtime.h"
#include <cutil.h>
#include <omp.h>
#include "cuda_pointer.h"

enum{
	MAX_GPU    = 4,
	MAX_CPU    = 8,
	NBODY_MAX  = (1<<18),
	NB_MAX     = 256,     // per block
	MAX_NB_BUF = (1<<18),
};

#include "gtx470.h"

#include "particle.h"

#define _out_

__global__ void kernel_jp_scatter(
		const int nj,
		const Jparticle jpsrc[],
		_out_ Jparticle jpdst[])
{
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < nj){
		const Jparticle jp = jpsrc[tid];
		jpdst[jp.addr] = jp;
	}
}

__global__ void kernel_predict(
		const int       nj,
		const float2    ti,
		const Jparticle jptcl[],
		_out_ Jppred    jpred[])
{
#if 0
	const int tid = threadIdx.x + blockDim.x * blockIdx.x;
	if(tid < nj){
		jpred[tid].predict(jptcl[tid], ti);
	}
#else
	const int tid = threadIdx.x;
	const int off = blockDim.x * blockIdx.x;
	const int nth = blockDim.x;
	__shared__ float4 sbuf[NTHREADS*5];
	Jparticle *sptcl = (Jparticle *)sbuf;
	Jppred    *spred = (Jppred    *)sbuf;

	{   // LOAD
		float4 *src = (float4 *)(jptcl + off);
		float4 *dst = (float4 *)(sptcl);
#pragma unroll
		for(int k=0; k<5; k++, src+=nth, dst+=nth){
			dst[tid] = src[tid];
		}
	}

	// Predict
	__syncthreads();
	Jppred pp;
	pp.predict(sptcl[tid], ti);
	__syncthreads();
	spred[tid] = pp;
	__syncthreads();

	{   // STORE
		float4 *src = (float4 *)(spred);
		float4 *dst = (float4 *)(jpred + off);
#pragma unroll
		for(int k=0; k<3; k++, src+=nth, dst+=nth){
			dst[tid] = src[tid];
		}
	}
#endif
}

#define INTERACTION Interaction_NB
__global__ void kernel_gravity(
		const int ni,
		const int nj,
		const Iparticle ipbuf[],
		const Jppred    jpbuf[],
		_out_ Force     fodev[][NJBLOCKS],
		_out_ int       nbbuf[][NJBLOCKS][NB_MAX],
		const bool      with_neib)
{
	int ibid = blockIdx.x;
	int jbid = blockIdx.y;
	int tid = threadIdx.x;
	int iaddr = tid + blockDim.x * ibid;
	int jstart = (nj * (jbid  )) / NJBLOCKS;
	int jend   = (nj * (jbid+1)) / NJBLOCKS;
	int *nbdst = nbbuf[iaddr][jbid];

	__shared__ Jppred jpshare[NJPSHRE]; // 32

	const Iparticle ip = ipbuf[iaddr];
	Force fo;
	fo.clear();
	
	if(with_neib){
		for(int j=jstart; j<jend; j+=NJPSHRE){
			const int jsize = NJPSHRE * Jppred::SIZE_F4; // 96

			__syncthreads();
			if(tid < jsize){ // 96 of 128
				float4 *src = (float4 *)(jpbuf + j);
				float4 *dst = (float4 *)(jpshare  );
				dst[tid] = src[tid];
			}
			if(tid+32 < jsize){ // for the case of 64 threads
				float4 *src = (float4 *)(jpbuf + j);
				float4 *dst = (float4 *)(jpshare  );
				dst[tid+32] = src[tid+32];
			}
			__syncthreads();

			if(jend-j < NJPSHRE){
#pragma unroll 4
				for(int jj=0; jj<jend-j; jj++){
					const Jppred jp = jpshare[jj];
					const Interaction_NB inter(ip, jp);
					inter.set_neib(nbdst[fo.num_neib % NB_MAX]);
					fo += inter;
				}
			}else{
#pragma unroll 32
				for(int jj=0; jj<NJPSHRE; jj++){
					const Jppred jp = jpshare[jj];
					const Interaction_NB inter(ip, jp);
					inter.set_neib(nbdst[fo.num_neib % NB_MAX]);
					fo += inter;
				}
			}
		}
	}else{ // no neib
		for(int j=jstart; j<jend; j+=NJPSHRE){
			const int jsize = NJPSHRE * Jppred::SIZE_F4; // 96

			__syncthreads();
			if(tid < jsize){ // 96 of 128
				float4 *src = (float4 *)(jpbuf + j);
				float4 *dst = (float4 *)(jpshare  );
				dst[tid] = src[tid];
			}
			if(tid+32 < jsize){ // for the case of 64 threads
				float4 *src = (float4 *)(jpbuf + j);
				float4 *dst = (float4 *)(jpshare  );
				dst[tid+32] = src[tid+32];
			}
			__syncthreads();

			if(jend-j < NJPSHRE){
#pragma unroll 4
				for(int jj=0; jj<jend-j; jj++){
					const Jppred jp = jpshare[jj];
					const Interaction inter(ip, jp);
					inter.set_neib(nbdst[fo.num_neib % NB_MAX]);
					fo += inter;
				}
			}else{
#pragma unroll 32
				for(int jj=0; jj<NJPSHRE; jj++){
					const Jppred jp = jpshare[jj];
					const Interaction inter(ip, jp);
					inter.set_neib(nbdst[fo.num_neib % NB_MAX]);
					fo += inter;
				}
			}
		}
	}

	if(iaddr < ni){
		fodev[iaddr][jbid] = fo;
	}
}

__global__ void kernel_reduce(
		const int ni,
		const Force fodev[][NJBLOCKS],
		_out_ Force fosum[]){
	const int xid = threadIdx.x;
	const int yid = threadIdx.y;
	const int bid = blockIdx.x;
	const int iaddr = yid + blockDim.y * bid;

	__shared__ Force fshare[NYREDUCE][NXREDUCE];
	if(xid < NJBLOCKS){
		fshare[yid][xid] = fodev[iaddr][xid];
	}else{
		fshare[yid][xid].clear();
	}
	Force *fs = fshare[yid];

	if(32 == NXREDUCE){
		if(xid < 16) fs[xid] += fs[xid + 16];
	}
	if(xid < 8) fs[xid] += fs[xid + 8];
	if(xid < 4) fs[xid] += fs[xid + 4];
	if(xid < 2) fs[xid] += fs[xid + 2];
	if(xid < 1) fs[xid] += fs[xid + 1];
	
	if(iaddr < ni && 0 == xid){
		fosum[iaddr] = fs[0];
	}
}

__global__ void kernel_gather_nb(
		const int   ni,
		const Force fodev[][NJBLOCKS],
		const int2  nbcnt[],
		const int   nbbuf[][NJBLOCKS][NB_MAX],
		_out_ int   nblst[])
{
	const int xid = threadIdx.x;
	const int yid = threadIdx.y;
	const int bid = blockIdx.x;
	const int iaddr = yid + blockDim.y * bid;
	if(iaddr >= ni) return;
	if(nbcnt[iaddr].x < 0) return; // overflow

	const int mynnb = (xid < NJBLOCKS) ? fodev[iaddr][xid].num_neib
	                                   : 0;

	// now performe prefix sum
	__shared__ int ishare[NYREDUCE][NXREDUCE];
	ishare[yid][xid] = mynnb;
	int *ish = ishare[yid];
	if(xid>=1)  ish[xid] += ish[xid-1];
	if(xid>=2)  ish[xid] += ish[xid-2];
	if(xid>=4)  ish[xid] += ish[xid-4];
	if(xid>=8)  ish[xid] += ish[xid-8];
	if(32 == NXREDUCE){
		if(xid>=16)  ish[xid] += ish[xid-16];
	}

	const int off = (xid == 0) ? 0 
	                           : ish[xid-1];
	int *nbdst = nblst + nbcnt[iaddr].y + off;
	if(xid < NJBLOCKS){
		for(int k=0; k<mynnb; k++){
			const int nbid = nbbuf[iaddr][xid][k];
			nbdst[k] = nbid;
		}
	}
}

class Resource{
private:
	bool   is_open;
	bool   predicted;
	bool   grav_called;
	bool   jp_flushed;
	int    gpid;
	int    njp_in_que;
	int    ni_save;
	float2 ti;

	cudaPointer<Iparticle>              ipbuf;
	cudaPointer<Jparticle>              jpbuf;
	cudaPointer<Jparticle>              jpque;
	cudaPointer<Jppred>                 jpred;
	cudaPointer <Force[NJBLOCKS]>       fodev;
	cudaPointer <Force>                 fosum;
	cudaPointer <int[NJBLOCKS][NB_MAX]> nbbuf;
	cudaPointer <int>                   nblst;
	cudaPointer <int2>                  nbcnt; // {num, off}

	void allocate(){
		ipbuf.allocate(NIMAX);
		jpbuf.allocate(NBODY_MAX);
		jpque.allocate(NBODY_MAX);
		jpred.allocate(NBODY_MAX);
		fodev.allocate(NIMAX);
		fosum.allocate(NIMAX);
		nbbuf.allocate(NIMAX);
		nblst.allocate(MAX_NB_BUF);
		nbcnt.allocate(NIMAX);
	}
	void free(){
		ipbuf.free();
		jpbuf.free();
		jpque.free();
		jpred.free();
		fodev.free();
		fosum.free();
		nbbuf.free();
		nblst.free();
		nbcnt.free();
	}
public:
	void set_gpid(const int id){
		gpid = id;
		int dev;
		hipGetDevice(&dev);
		assert(dev == gpid);
	}
	void open(){
		assert(!is_open);
		allocate();
		is_open     = true;
		njp_in_que  = 0;
		predicted   = false;
		grav_called = false;
		jp_flushed  = false;
	}
	void close(){
		assert(is_open);
		free();
		is_open = false;
	}
	void set_ip(
			const int    ni,
			const double pos[][3],
			const double vel[][3],
			const double eps2[],
			const double h2  [],
			const int    id  [])
	{
		assert(is_open);
		assert(ni <= NIMAX);
		
		for(int i=0; i<ni; i++){
			ipbuf[i].read(pos[i], vel[i], eps2[i], h2[i], id[i]);
		}
		ipbuf.htod(ni);
		ni_save = ni;
	}
	void push_jp(
			const double pos [3],
			const double vel [3],
			const double acc2[3],
			const double jrk6[6],
			const double mass,
			const double tj,
			const int    id,
			const int    addr)
	{
		assert(is_open);
		assert(addr < NBODY_MAX);

		jpque[njp_in_que].read(pos, vel, acc2, jrk6, mass, tj, id, addr);
		njp_in_que++;
		jp_flushed = false;
	}
	void transter_jp(){
		assert(is_open);

		const int njq = njp_in_que;
		jpque.htod(njq);
		njp_in_que = 0;
		const int Blocks = 1 + (njq-1)/NTHSCAT;
		kernel_jp_scatter <<< Blocks, NTHSCAT >>>
			(njq, jpque, jpbuf);
		jp_flushed = true;
		predicted  = false;
	}
	void set_ti(const double dbl_ti){
		assert(is_open);

		ti = float2_split(dbl_ti);
		predicted = false;
	}
	void predict_all(const int nj){
		assert(is_open);
		
		const int Blocks = 1 + (nj-1)/NTHREADS;
		kernel_predict <<< Blocks, NTHREADS >>>
			(nj, ti, jpbuf, jpred);
		predicted = true;
	}
	void launch_gravity(
			const int  ni,
			const int  nj,
			const bool with_neib)
	{
		assert(is_open);
		assert(ni == ni_save);
		assert(ni <= NIMAX);
		assert(nj < NBODY_MAX);

		if(!jp_flushed) transter_jp();
		if(!predicted ) predict_all(nj);
		if(ni <= 64){
			dim3 grid   ( 1, NJBLOCKS, 1);
			dim3 threads(64,        1, 1);
			kernel_gravity <<< grid, threads >>>
				(ni, nj, ipbuf, jpred, fodev, nbbuf, with_neib);
		}else{
			const int niblocks = 1 + (ni-1) / NTHREADS;
			dim3 grid   (niblocks, NJBLOCKS, 1);
			dim3 threads(NTHREADS,        1, 1);
			kernel_gravity <<< grid, threads >>>
				(ni, nj, ipbuf, jpred, fodev, nbbuf, with_neib);
		}
		grav_called = true;
	}
	void get_force(
			const int    ni,
			_out_ double acc   [][3],
			_out_ double jrk   [][3],
			_out_ double pot   [],
			_out_ int    nnb_id[])
	{
		assert(is_open);
		assert(grav_called);
		assert(ni == ni_save);
		assert(ni <= NIMAX);

		const int ni8 = 1 + (ni-1) / NYREDUCE;
		dim3 grid   (ni8, 1, 1);
		dim3 threads(NXREDUCE, NYREDUCE, 1);
		kernel_reduce <<< grid, threads >>>
			(ni, fodev, fosum);
		fosum.dtoh(ni);
		grav_called = false;

		for(int i=0; i<ni; i++){
			fosum[i].write(acc[i], jrk[i], pot[i], nnb_id[i], nbcnt[i].x);
		}
	}
	void receive_neighbor_list(){
		assert(is_open);

		const int ni = ni_save;
		int nbsum = 0;
		for(int i=0; i<ni; i++){
			nbcnt[i].y = nbsum;
			if(nbcnt[i].x >= 0) nbsum += nbcnt[i].x;
		}
		assert(nbsum <= MAX_NB_BUF);
		nbcnt.htod(ni);

		const int ni8 = 1 + (ni-1) / NYREDUCE;
		dim3 grid   (ni8, 1, 1);
		dim3 threads(NXREDUCE, NYREDUCE, 1);
		kernel_gather_nb <<< grid, threads >>>
			(ni, fodev, nbcnt, nbbuf, nblst);
		nblst.dtoh(nbsum);
	}
	void get_neighbor_list(
			const int ipipe,
			const int maxlen,
			_out_ int *num_neib,
			_out_ int list[])
	{
		assert(is_open);
		assert(ipipe < ni_save);

		const int nnb = nbcnt[ipipe].x;
		const int off = nbcnt[ipipe].y;
		const int *src = &nblst[off];
		if(nnb > 0 && maxlen >= nnb){
			for(int k=0; k<nnb; k++){
				list[k] = src[k];
			}
			*num_neib = nnb;
		}else{
			*num_neib = -1;
		}
	}

	void DEBUG_read_pred(
			const int    nj,
			const int    addr,
			_out_ double pos [3],
			_out_ double vel [3],
			_out_ double mass[1],
			_out_ int    id  [1])
	{
		jpred.dtoh(nj);
		const Jppred &p = jpred[addr];
		for(int k=0; k<3; k++){
			pos[k] = p.pos[k].x + p.pos[k].y;
			vel[k] = p.vel[k];
		}
		mass[0] = p.mass;
		id  [0] = p.id;
	}

};

static Resource resource[MAX_GPU];
static int numGPU, numCPU;
static bool initialized = false;

static void lib_initialize(){
	if(initialized) return;
	initialized = true;

	assert(NXREDUCE >= NJBLOCKS);
	assert(NXREDUCE <= 32);
	assert(sizeof(Jppred) % sizeof(float4) == 0);
	assert(sizeof(Jppred) / sizeof(float4) == Jppred::SIZE_F4);
	assert(NJPSHRE * Jppred::SIZE_F4 <= NTHREADS);

	int devid[MAX_GPU];
	hipGetDeviceCount(&numGPU);
	assert(numGPU <= MAX_GPU);
	char *gpu_list = getenv("GPU_LIST");
	if(gpu_list){
		// get GPU list from environment variable
		numGPU = 0;
		char *p = strtok(gpu_list, " ");
		while(p){
			devid[numGPU++] = atoi(p);
			p = strtok(NULL, " ");
			assert(numGPU <= MAX_GPU);
		}
	}else{
		// use all GPUs
		for(int i=0; i<numGPU; i++){
			devid[i] = i;
		}
	}

	// numGPU = 1;
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid == 0) numCPU = omp_get_num_threads();
	}
	assert(numCPU <= MAX_CPU);
	assert(numGPU <= numCPU);
#pragma omp parallel
	{
		int tid = omp_get_thread_num();
		if(tid < numGPU){
			hipSetDevice(devid[tid]);
			resource[tid].set_gpid(devid[tid]);
		}
	}
	fprintf(stderr, "***********************\n");
	fprintf(stderr, "Initializing Yebisu/G6 library\n");
	fprintf(stderr, "#CPU %d, #GPU %d\n", numCPU, numGPU);
	fprintf(stderr, " device:");
	for(int i=0; i<numGPU; i++){
		fprintf(stderr, " %d", devid[i]);
	}
	fprintf(stderr, "\n");
	fprintf(stderr, "***********************\n");

#if 1
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_jp_scatter), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_predict),    hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_gravity),    hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_reduce),     hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_gather_nb),  hipFuncCachePreferL1);
#endif
}

#include "yebisu_g6.h"
extern "C"{
	void yebisu_g6_open (const int gpid){
		lib_initialize();
		resource[gpid].open();
	}
	void yebisu_g6_close(const int gpid){
		lib_initialize();
		resource[gpid].close();
	}
	void yebisu_g6_set_ip(
			const int    gpid,
			const int    ni,
			const double pos[][3],
			const double vel[][3],
			const double eps2[],
			const double h2  [],
			const int    id  [])
	{
		lib_initialize();
		resource[gpid].set_ip(ni, pos, vel, eps2, h2, id);
	}
	void yebisu_g6_push_jp(
			const int    gpid,
			const double pos [3],
			const double vel [3],
			const double acc2[3],
			const double jrk6[6],
			const double mass,
			const double tj,
			const int    id,
			const int    addr)
	{
		lib_initialize();
		resource[gpid].push_jp( pos, vel, acc2, jrk6, mass, tj, id, addr);
	}
	void yebisu_g6_transfer_jp(const int gpid){
		lib_initialize();
		resource[gpid].transter_jp();
	}
	void yebisu_g6_set_ti(
			const int    gpid,
			const double ti)
	{
		lib_initialize();
		resource[gpid].set_ti(ti);
	}
	void yebisu_g6_predict_all(
			const int gpid,
			const int nj)
	{
		lib_initialize();
		resource[gpid].predict_all(nj);
	}
	void yebisu_g6_launch_gravity(
			const int gpid,
			const int ni,
			const int nj,
			const int with_neib)
	{
		lib_initialize();
		resource[gpid].launch_gravity(ni, nj, bool(with_neib));
	}
	void yebisu_g6_get_force(
			const int    gpid,
			const int    ni,
			_out_ double acc   [][3],
			_out_ double jrk   [][3],
			_out_ double pot   [],
			_out_ int    nnb_id[])
	{
		lib_initialize();
		resource[gpid].get_force(ni, acc, jrk, pot, nnb_id);
	}
	void yebisu_g6_receive_neighbor_list(const int gpid){
		lib_initialize();
		resource[gpid].receive_neighbor_list();
	}
	void yebisu_g6_get_neighbor_list(
			const int gpid,
			const int ipipe,
			const int maxlen,
			_out_ int *num_neib,
			_out_ int list[])
	{
		lib_initialize();
		resource[gpid].get_neighbor_list(ipipe, maxlen, num_neib, list);
	}

	void yebisu_g6_DEBUG_read_pred(
			const int    gpid,
			const int    nj,
			const int    addr,
			_out_ double pos [3],
			_out_ double vel [3],
			_out_ double mass[1],
			_out_ int    id  [1])
	{
		resource[gpid].DEBUG_read_pred(nj, addr, pos, vel, mass, id);
	}

	int yebisu_g6_get_nimax(){
		return NIMAX;
	}

	int yebisu_g6_get_njmax(){
		return NBODY_MAX;
	}
}
